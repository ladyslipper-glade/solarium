#include "hip/hip_runtime.h"
/*! \file    Object.c
 *  \brief   Implementation of object data arrays.
 *  \author  Peter C. Chapin <pchapin@vtc.edu>
 */

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#include "global.h"
#include "Initialize.h"
#include "CUDAVector3.h"

Object         *object_array;
ObjectDynamics *current_dynamics;
ObjectDynamics *next_dynamics;

#define BLOCK_SIZE 256

__global__ void do_calculations(
    Object *object_array, ObjectDynamics *current_dynamics, ObjectDynamics *next_dynamics )
{
    int object_i = blockIdx.x * blockDim.x + threadIdx.x;

    // For each object...
    if( object_i < OBJECT_COUNT ) {
        Vector3 total_force = { 0.0F, 0.0F, 0.0F };

        const Vector3 pos_i = current_dynamics[object_i].position;
        const float mass_i = object_array[object_i].mass;

        // Shared memory is shared between all the threads of a block.
        __shared__ Vector3 shared_position[BLOCK_SIZE];
        __shared__ float shared_mass[BLOCK_SIZE];

        int blocks = (OBJECT_COUNT + BLOCK_SIZE - 1) / BLOCK_SIZE;

        // We consider a block's worth of "other" objects at a time.
        for (unsigned block_i = 0; block_i < blocks-1; block_i++) {

            // Populate the shared information (essentially, cache it).
            // Since everythread in the block is doing this, the entire shared arrays are filled.
            shared_position[threadIdx.x] = current_dynamics[block_i * blockDim.x + threadIdx.x].position;
            shared_mass[threadIdx.x] = object_array[block_i * blockDim.x + threadIdx.x].mass;

            // Wait for all the threads in the block to finish populating the shared arrays.
            __syncthreads();

            // Consider interactions with all other objects...
            // Here object_i is an overall object index, but object_j is a per-block object index.
            for (int object_j = 0; object_j < BLOCK_SIZE; ++object_j) {
                if( object_i == block_i * blockDim.x + object_j ) continue;

                Vector3 displacement = cuda_v3_subtract( shared_position[object_j], pos_i );
                const float distance_squared = cuda_magnitude_squared( displacement );
                const float distance = sqrt( distance_squared );
                const float t1 = mass_i / distance;
                const float t2 = shared_mass[object_j] / distance;
                //float force_magnitude =
                //    ( G * object_array[object_i].mass * object_array[object_j].mass ) / distance_squared;
                const float force_magnitude = ( G * t1 ) * t2;
                const Vector3 force = cuda_v3_multiply( (force_magnitude / distance ), displacement );
                total_force = cuda_v3_add( total_force, force );
            }

            
            // Wait for all threads in the block to finish computations before loading the next block.
            __syncthreads();
        }

        // Now we must deal with the last partial block...

        if ((blocks-1) * blockDim.x + threadIdx.x < OBJECT_COUNT) {
            shared_position[threadIdx.x] =
                current_dynamics[(blocks-1) * blockDim.x + threadIdx.x].position;
            shared_mass[threadIdx.x] =
                object_array[(blocks-1) * blockDim.x + threadIdx.x].mass;
        }
        __syncthreads();
        for (int object_j = 0; object_j < OBJECT_COUNT - BLOCK_SIZE * (blocks-1); ++object_j) {
            if( object_i == (blocks-1) * blockDim.x + object_j ) continue;

            Vector3 displacement = cuda_v3_subtract( shared_position[object_j], pos_i );
            float distance_squared = cuda_magnitude_squared( displacement );
            float distance = sqrt( distance_squared );
            float t1 = mass_i / distance;
            float t2 = shared_mass[object_j] / distance;
            //float force_magnitude =
            //    ( G * object_array[object_i].mass * object_array[object_j].mass ) / distance_squared;
            float force_magnitude = ( G * t1 ) * t2;
            Vector3 force = cuda_v3_multiply( (force_magnitude / distance ), displacement );
            total_force = cuda_v3_add( total_force, force );
        }

        // Total force on object_i is now known. Compute acceleration, velocity and position.
        Vector3 acceleration   = cuda_v3_divide( total_force, mass_i );
        Vector3 delta_v        = cuda_v3_multiply( TIME_STEP, acceleration );
        Vector3 delta_position = cuda_v3_multiply( TIME_STEP, current_dynamics[object_i].velocity );

        // Update the dynamics arrays.
        next_dynamics[object_i].velocity =
            cuda_v3_add( current_dynamics[object_i].velocity, delta_v );

        next_dynamics[object_i].position =
            cuda_v3_add( current_dynamics[object_i].position, delta_position );
    }
}


void cuda_time_step(
    Object         *dev_object_array,
    ObjectDynamics *dev_current_dynamics,
    ObjectDynamics *dev_next_dynamics )
{
    int block_count =
        (OBJECT_COUNT % BLOCK_SIZE == 0) ? OBJECT_COUNT/BLOCK_SIZE
                                         : OBJECT_COUNT/BLOCK_SIZE + 1;

    // Strictly speaking we don't need to copy the current_dynamics array back and forth. The
    // computed values could be left on the device during the whole computation and copied back
    // only at the end. [But be careful: you still need to swap the dynamics arrays on the
    // device between time steps]. However, experiments show the overhead of this copying is
    // small compared to the time required to do the computation above.

    hipMemcpy( dev_current_dynamics,
                current_dynamics,
                OBJECT_COUNT * sizeof(ObjectDynamics),
                hipMemcpyHostToDevice );

    size_t size_shared = BLOCK_SIZE * (sizeof(next_dynamics[0].position) + sizeof(object_array[0].mass));
    do_calculations<<<block_count, BLOCK_SIZE, size_shared>>>(
        dev_object_array, dev_current_dynamics, dev_next_dynamics );

    // Notice that we copy the next_dynamics on the device to the current_dynamics on the host.
    // This means we don't have to worry about swapping the arrays explicitly.
    hipMemcpy( current_dynamics,
                dev_next_dynamics,
                OBJECT_COUNT * sizeof(ObjectDynamics),
                hipMemcpyDeviceToHost );
}


void dump_dynamics( )
{
    for( int object_i = 0; object_i < OBJECT_COUNT; ++object_i ) {
        printf( "%4d: x = %11.3E, y = %11.3E, z = %11.3E\n", object_i,
                current_dynamics[object_i].position.x/AU,
                current_dynamics[object_i].position.y/AU,
                current_dynamics[object_i].position.z/AU );
    }
}
