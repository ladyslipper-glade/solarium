#include "hip/hip_runtime.h"
/*! \file    Object.c
    \brief   Implementation of object data arrays.
    \author  Peter C. Chapin <PChapin@vtc.vsc.edu>

*/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>

#include "global.h"
#include "Initialize.h"
#include "CUDAVector3.h"

Object         *object_array;
ObjectDynamics *current_dynamics;
ObjectDynamics *next_dynamics;


__global__ void do_calculations(
    Object *object_array, ObjectDynamics *current_dynamics, ObjectDynamics *next_dynamics )
{
    int object_i = blockIdx.x*blockDim.x + threadIdx.x;

    // For each object...
    if( object_i < OBJECT_COUNT ) {
        Vector3 total_force = { 0.0F, 0.0F, 0.0F };

        // Consider interactions with all other objects...
        for( int object_j = 0; object_j < OBJECT_COUNT; ++object_j ) {
            if( object_i == object_j ) continue;

            Vector3 displacement = cuda_v3_subtract(
                current_dynamics[object_j].position, current_dynamics[object_i].position );
            float distance_squared = cuda_magnitude_squared( displacement );
            float distance = sqrt( distance_squared );
            float t1 = object_array[object_i].mass / distance;
            float t2 = object_array[object_j].mass / distance;
            //float force_magnitude =
            //    ( G * object_array[object_i].mass * object_array[object_j].mass ) / distance_squared;
            float force_magnitude = ( G * t1 ) * t2;
            Vector3 force = cuda_v3_multiply( (force_magnitude / distance ), displacement );
            total_force = cuda_v3_add( total_force, force );
        }

        // Total force on object_i is now known. Compute acceleration, velocity and position.
        Vector3 acceleration =
            cuda_v3_divide( total_force, object_array[object_i].mass );

        Vector3 delta_v =
            cuda_v3_multiply( TIME_STEP, acceleration );

        Vector3 delta_position =
            cuda_v3_multiply( TIME_STEP, current_dynamics[object_i].velocity );

        // Update the dynamics arrays.
        next_dynamics[object_i].velocity =
            cuda_v3_add( current_dynamics[object_i].velocity, delta_v );

        next_dynamics[object_i].position =
            cuda_v3_add( current_dynamics[object_i].position, delta_position );
    }
}


void cuda_time_step(
    Object         *dev_object_array,
    ObjectDynamics *dev_current_dynamics,
    ObjectDynamics *dev_next_dynamics )
{
    int block_size = 256;  // Number of threads per block.
    int block_count =
        (OBJECT_COUNT % block_size == 0) ? OBJECT_COUNT/block_size
                                         : OBJECT_COUNT/block_size + 1;

    // Strictly speaking we don't need to copy the current_dynamics array back and forth.
    // The computed values could be left on the device during the whole computation and copied
    // back only at the end. However, experiments show the overhead of this copying is small
    // compared to the time required to do the computation above.

    hipMemcpy( dev_current_dynamics,
                current_dynamics,
                OBJECT_COUNT*sizeof(ObjectDynamics),
                hipMemcpyHostToDevice );

    do_calculations<<<block_count, block_size>>>(
        dev_object_array, dev_current_dynamics, dev_next_dynamics );

    hipMemcpy( current_dynamics,
                dev_next_dynamics,
                OBJECT_COUNT*sizeof(ObjectDynamics),
                hipMemcpyDeviceToHost );
}


void dump_dynamics( )
{
    for( int object_i = 0; object_i < OBJECT_COUNT; ++object_i ) {
        printf( "%4d: x = %10.3E, y = %10.3E, z = %10.3E\n", object_i,
                current_dynamics[object_i].position.x/AU,
                current_dynamics[object_i].position.y/AU,
                current_dynamics[object_i].position.z/AU );
    }
}
