/*! \file    main.c
 *  \brief   Main program of the CUDA solar system simulator.
 *  \author  Peter C. Chapin <pchapin@vtc.edu>
 *
 * LICENSE
 *
 * This program is free software; you can redistribute it and/or modify it under the terms of
 * the GNU General Public License as published by the Free Software Foundation; either version 2
 * of the License, or (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful, but WITHOUT ANY WARRANTY;
 * without even the implied warranty of MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See
 * the GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License along with this program; if
 * not, write to the Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA
 * 02111-1307 USA
 */

#include <stdio.h>
#include <stdlib.h>

#include "global.h"
#include "Initialize.h"
#include "Timer.h"

#define STEPS_PER_YEAR 8766  // Number of hours in a year.

int main( int argc, char **argv )
{
    Timer stopwatch;
    long long total_steps = 0;
    int total_years       = 0;
    int return_code       = EXIT_SUCCESS;

    Object         *dev_object_array;       // Pointer to object array on GPU.
    ObjectDynamics *dev_current_dynamics;   // Pointer to current dynamics array on GPU.
    ObjectDynamics *dev_next_dynamics;      // Pointer to next dynamics array on GPU.

    initialize_object_arrays( );
    printf( "START position\n" );
    dump_dynamics( );
    Timer_initialize( &stopwatch );
    Timer_start( &stopwatch );

    // Allocate memory on GPU.
    // TODO: Add error checking!
    hipMalloc( (void**)&dev_object_array, OBJECT_COUNT*sizeof(Object) );
    hipMalloc( (void**)&dev_current_dynamics, OBJECT_COUNT*sizeof(ObjectDynamics) );
    hipMalloc( (void**)&dev_next_dynamics, OBJECT_COUNT*sizeof(ObjectDynamics) );

    // Copy the object array to the GPU. This never changes so this only needs to be done once.
    hipMemcpy( dev_object_array,
                object_array,
                OBJECT_COUNT*sizeof(Object),
                hipMemcpyHostToDevice );

    // Maybe copy the dynamics arrays once and leave them on the device...? See note in Object.cu.
    //
    //hipMemcpy( dev_current_dynamics,
    //            current_dynamics,
    //            OBJECT_COUNT*sizeof(ObjectDynamics),
    //            hipMemcpyHostToDevice );

    while (1) {
        cuda_time_step( dev_object_array, dev_current_dynamics, dev_next_dynamics );
        total_steps++;

        // Print out a message after 100 steps just to give the user something to see.
        if( total_steps % 100 == 0 )
            fprintf( stderr, "STEP %4lld\n", total_steps );

        if( total_steps % STEPS_PER_YEAR == 0 ) {
            total_years++;
            if( total_years % 10 == 0 ) {
                fprintf( stderr, "Years simulated = %d\r", total_years );
                fflush( stderr );
            }

            // For now, stop the simulation after 1 year.
            if( total_years == 1 ) break;
        }
    }

    //hipMemcpy( current_dynamics,
    //            dev_next_dynamics,
    //            OBJECT_COUNT*sizeof(ObjectDynamics),
    //            hipMemcpyDeviceToHost );

    // Deallocate memory on GPU.
    hipFree( dev_object_array );
    hipFree( dev_current_dynamics );
    hipFree( dev_next_dynamics );

    Timer_stop( &stopwatch );
    printf( "\nEND position\n" );
    dump_dynamics( );
    printf( "Time elapsed = %ld milliseconds\n", Timer_time( &stopwatch ) );

    return return_code;
}
