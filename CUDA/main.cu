/*! \file    main.c
 *  \brief   Main program of the CUDA solar system simulator.
 *  \author  Peter Chapin <peter.chapin@vermontstate.edu>
 */

#include <stdio.h>
#include <stdlib.h>

#include "global.h"
#include "Initialize.h"
#include "Timer.h"

#define STEPS_PER_YEAR 8766  // Number of hours in a year.

int main( int argc, char **argv )
{
    Timer stopwatch;
    long long total_steps = 0;
    int total_years       = 0;
    int return_code       = EXIT_SUCCESS;

    Object         *dev_object_array;       // Pointer to object array on GPU.
    ObjectDynamics *dev_current_dynamics;   // Pointer to current dynamics array on GPU.
    ObjectDynamics *dev_next_dynamics;      // Pointer to next dynamics array on GPU.

    initialize_object_arrays( );
    printf( "START position\n" );
    dump_dynamics( );
    Timer_initialize( &stopwatch );
    Timer_start( &stopwatch );

    // Allocate memory on GPU.
    // TODO: Add error checking!
    hipMalloc( (void**)&dev_object_array, OBJECT_COUNT * sizeof(Object) );
    hipMalloc( (void**)&dev_current_dynamics, OBJECT_COUNT * sizeof(ObjectDynamics) );
    hipMalloc( (void**)&dev_next_dynamics, OBJECT_COUNT * sizeof(ObjectDynamics) );

    // Copy the object array to the GPU. This never changes so this only needs to be done once.
    hipMemcpy( dev_object_array,
                object_array,
                OBJECT_COUNT*sizeof(Object),
                hipMemcpyHostToDevice );

    // Maybe copy the dynamics arrays once and leave them on the device...? See note in Object.cu.
    //
    //hipMemcpy( dev_current_dynamics,
    //            current_dynamics,
    //            OBJECT_COUNT*sizeof(ObjectDynamics),
    //            hipMemcpyHostToDevice );

    while( 1 ) {
        cuda_time_step( dev_object_array, dev_current_dynamics, dev_next_dynamics );
        total_steps++;

        // Print out a message after 100 steps just to give the user something to see.
        if( total_steps % 100 == 0 )
            fprintf( stderr, "STEP %4lld\n", total_steps );

        if( total_steps % STEPS_PER_YEAR == 0 ) {
            total_years++;
            if( total_years % 10 == 0 ) {
                fprintf( stderr, "Years simulated = %d\r", total_years );
                fflush( stderr );
            }

            // For now, stop the simulation after 1 year.
            if( total_years == 1 ) break;
        }
    }

    //hipMemcpy( current_dynamics,
    //            dev_next_dynamics,
    //            OBJECT_COUNT*sizeof(ObjectDynamics),
    //            hipMemcpyDeviceToHost );

    // Deallocate memory on GPU.
    hipFree( dev_object_array );
    hipFree( dev_current_dynamics );
    hipFree( dev_next_dynamics );

    Timer_stop( &stopwatch );
    printf( "\nEND position\n" );
    dump_dynamics( );
    printf( "Time elapsed = %ld milliseconds\n", Timer_time( &stopwatch ) );

    return return_code;
}
